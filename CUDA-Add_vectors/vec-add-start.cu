// Adapted from NVIDIA's examples, scaler value with one block, one thread

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) { 
  *c = *a + *b;
}

__host__ void usage() {
	fprintf(stderr, "Usage: vec-add-start a b\n");
	exit(1);
}


int main(int argc, char** argv) {
  int a, b, c; // host copies of a, b, c 
  int *d_a, *d_b, *d_c; // device copies of a, b, c 
  int size = sizeof(int);
               
  // Setup input values  
	if (argc != 3) 
		usage();
	if (sscanf(argv[1], "%d", &a) != 1) 
		usage();
	if (sscanf(argv[2], "%d", &b) != 1) 
		usage();

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size); 
  hipMalloc((void **)&d_b, size); 
  hipMalloc((void **)&d_c, size);

  // Copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice); 
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  add<<< 1, 1 >>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  fprintf(stdout, "%d + %d = %d\n", a, b, c); 

  // Cleanup
  hipFree(d_a); 
  hipFree(d_b); 
  hipFree(d_c);
  
  return(0);
}
